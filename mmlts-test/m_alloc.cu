#include "hip/hip_runtime.h"
#include "m_alloc.h"

__global__ void InitializeMatrix_kernel(float *matrix, int rows, int columns, int seed)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < rows && j < columns) {
		int offset = i + j * rows;

		hiprandState_t state;
		hiprand_init(seed, offset, 0, &state);
		matrix[offset] = hiprand_uniform(&state);
	}
}

__global__ void NormalizeMatrix_kernel(float *matrix, int row, int column)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= row)
		return;

	float sum = 0;
	for (int j = 0; j < column; j++) {
		float item = matrix[i + j * row];
		sum += item * item;
	}
	sum = sqrt(sum);
	for (int j = 0; j < column; j++) {
		float item = matrix[i + j * row];
		matrix[i + j * row] = item / sum;
	}
}

hipError_t NormalizeMatrix(float *matrix, int rows, int columns)
{
	dim3 block(256);
	dim3 grid((rows + 256 - 1) / 256);

	NormalizeMatrix_kernel<<< grid, block >>>(matrix, rows, columns);

	return hipGetLastError();
}

hipError_t InitializeMatrix(float *matrix, int rows, int columns)
{
	dim3 block(16, 16);
	dim3 grid(
		(rows + block.x - 1) / block.x,
		(columns + block.y - 1) / block.y
	);

	InitializeMatrix_kernel<<< grid, block >>>(matrix, rows, columns, std::rand());

	return hipGetLastError();
}

hipError_t AllocateMatrix(float **matrix, int rows, int columns, bool initialized)
{
	hipError_t result;

	size_t sizeof_matrix = sizeof(float) * rows * columns;

	// Allocate device memory.
	result = hipMallocManaged((void **)(matrix), sizeof_matrix);
	//result = hipMalloc((void **)(matrix), sizeof_matrix);
	check_return(
		result, hipSuccess,
		"Failed to allocate matrix: %s\n",
		hipGetErrorString(result)
	);

	// Clear the allocation.
	result = hipMemset(*matrix, 0, sizeof_matrix);
	check_return(
		result, hipSuccess,
		"Failed to clear matrix device memory: %s\n",
		hipGetErrorString(result)
	);

	if (!initialized)
		return result;

	// Initialize matrix elements to arbitrary small integers.
	result = InitializeMatrix(*matrix, rows, columns);
	check_return(
		result, hipSuccess,
		"Failed to initialize matrix: %s\n",
		hipGetErrorString(result)
	);

	return result;
}
