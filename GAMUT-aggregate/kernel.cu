#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ __launch_bounds__(
    standard::Count_Thr_Block_Tot) void standard::compute(Params param) {
  __shared__ uint8_t SharedMemoryBase[sizeof(SharedMemory)];

  SharedMemory *smem = (SharedMemory *)SharedMemoryBase;

  int thread_idx = threadIdx.x;
  int warp_idx = thread_idx / Count_Thr_Warp_Tot;
  int lane_idx = thread_idx % Count_Thr_Warp_Tot;

  idx id = {int(blockIdx.x),
            int(blockIdx.y),
            int(blockIdx.z),
            warp_idx % Count_Warp_Block_Row,
            warp_idx / Count_Warp_Block_Row,
            lane_idx % Count_Thr_Warp_Row,
            lane_idx / Count_Thr_Warp_Row,
            thread_idx};

  Array<Type, Size_Thr_Row> in_regs_row[Num_Inputs_Row];
  Array<Type, Size_Thr_Col> in_regs_col[Num_Inputs_Col];
  Array<Type, Size_Thr_Tot> *in_regs_tot = nullptr;

  InIterator in_iter(in_regs_row, in_regs_col, in_regs_tot, smem->in_smem,
                     param, id);

  Array<Type, Size_Thr_Tot> out_reg;
  out_reg.fill(0);

  int size_K_ =
      min(param.size_K, param.size_p.kDepth - id.block_k * param.size_K);
  int k_iterations = (size_K_ + Depth_Block - 1) / Depth_Block;

  MMLT_LOOP
  for (; k_iterations > 0; k_iterations--) {
    in_iter.load_next_smem();

    __syncthreads();

    MMLT_UNROLL
    for (int k = 0; k < Depth_Block; k++) {
      in_iter.load_next_reg();
      mmlt_op(out_reg, in_regs_row, in_regs_col, in_regs_tot);
    }
    in_iter.reset_smem_offset();

    __syncthreads();
  }

  if (param.size_K == param.size_p.kDepth) {
    MMLT_UNROLL
    for (int i = 0; i < Size_Thr_Row; i++) {
      for (int j = 0; j < Size_Thr_Col; j++) {
        coda_op(out_reg.storage[i * Size_Thr_Col + j]);
      }
    }
  }

  OutIterator out_iter(out_reg, in_regs_row, in_regs_col, in_regs_tot,
                       smem->out_smem, param, id);
  MMLT_UNROLL
  for (int i = 0; i < Size_Thr_Row; i++) {
    out_iter.store_next_reg();
    __syncthreads();

    out_iter.store_next_smem();
    __syncthreads();
  }
}

__global__ __launch_bounds__(
    standard::Reduce_Row
        *standard::Reduce_Col) void standard::reduce(Params param) {
  coord<2> offset = {int(blockIdx.x) * Reduce_Row + int(threadIdx.y),
                     int(blockIdx.y) * Reduce_Col + int(threadIdx.x)};

  if (offset.x >= param.size_p.kRow || offset.y >= param.size_p.kColumn) {
    return;
  }

  Type accum = 0;
  Type thread_work[Reduce_Inner_Iter];

  int off = offset.x * param.size_p.kColumn + offset.y;
  Type *work_base = (Type *)param.work + off;
  Type *output_base = (Type *)param.output + off;

  off = param.size_p.kRow * param.size_p.kColumn;

  MMLT_LOOP
  for (int k = 0; k < param.n_partition; k += Reduce_Inner_Iter) {
    MMLT_UNROLL
    for (int i = 0; i < Reduce_Inner_Iter; i++) {
      if (k + i < param.n_partition) {
        thread_work[i] = *work_base;
        work_base += off;
      }
    }

    MMLT_UNROLL
    for (int i = 0; i < Reduce_Inner_Iter; i++) {
      if (k + i < param.n_partition) {
        reduce_op(accum, thread_work[i]);
      }
    }
  }

  coda_op(accum);

  *output_base = accum;
}